#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <fftw3.h>

#include "hip/hip_runtime_api.h"

#include "fft_helper.cuh"

using std::vector;


// Utility method for computing the array of frequencies given the number of frequency bins M.
vector<float> getFreq(float df, int M) {
    vector<float> freq(M);
    for (int i=0; i < freq.size(); i++) {
        freq[i] = df * (-M/2.0f +i);
    }
    return freq;
}


// Compute parameters for gridding.
struct Param computeGridParams(const int M, const float eps) {
    // Choosing the interpolation Gaussian kernel parameters based on Dutt & Rohklin (1993)
    struct Param param;
    param.tau = 0.0f;
    // Oversampling ratio. ratio=3 gives higher accuracy.
    int ratio = 3;
    param.Msp = static_cast<int>(-std::log(eps) / (PI * (ratio - 1) / (ratio - 0.5f)) + 0.5f);
    param.Mr = std::max(2 * param.Msp, ratio * M);
    param.tau = PI * (param.Msp/ (ratio * (ratio - 0.5f))) / powf(M, 2.0f);
    return param;
}


// CPU FFT on a regular grid using FFTW
vector<Complex> fftCpu(vector<float> inp, const int iflag) {
    int n = inp.size();
    vector<Complex> out(n);
    fftwf_plan p;
    fftwf_complex *inCopied;
    fftwf_complex *outTemp;
    inCopied = (fftwf_complex*) fftw_malloc(sizeof(fftwf_complex) * n);
    outTemp = (fftwf_complex*) fftw_malloc(sizeof(fftwf_complex) * n);
    for (int i=0; i< n; i++) {
        inCopied[i][0] = inp[i];
        inCopied[i][1] = 0.0f;
    }

    if (iflag < 0) {
        p =  fftwf_plan_dft_1d(n, inCopied, outTemp, FFTW_FORWARD, FFTW_ESTIMATE);
    } else {
        // Cast inp in fftw_complex and use regular fftw
        p =  fftwf_plan_dft_1d(n, inCopied, outTemp, FFTW_BACKWARD, FFTW_ESTIMATE);
    }

    fftwf_execute(p);

    for (int i=0; i < n; i++) {
        out[i] = Complex(outTemp[i][0], outTemp[i][1]);
    }

    fftw_free(inCopied);
    fftw_free(outTemp);
    fftwf_destroy_plan(p);
    return out;
}


void callCufft(hipfftComplex *dev_in, hipfftComplex *dev_out, int n, const int iflag) {
    hipfftHandle plan;
    int batch = 1;
    CUFFT_CALL(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));
    if (iflag < 0) {
        CUFFT_CALL(hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_FORWARD));
    } else {
        CUFFT_CALL(hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_BACKWARD));
    }
    CUFFT_CALL(hipfftDestroy(plan));
}

// GPU FFT for testing CUFFT against FFTW
vector<Complex> fftGpu(vector<float> inp, const int iflag) {
    int n = inp.size();
    vector<Complex> out(n);
    hipfftComplex complex_in[n];
    hipfftComplex *dev_in;
    hipfftComplex *dev_out;
    CUDA_CALL(hipMalloc((void **) &dev_in, n * sizeof(hipfftComplex)));
    CUDA_CALL(hipMalloc((void **) &dev_out, n * sizeof(hipfftComplex)));

    for (int i=0;i<inp.size(); ++i) {
        complex_in[i].x = inp[i];
        complex_in[i].y = 0.0f;
    }

    CUDA_CALL(hipMemcpy(dev_in, complex_in, n * sizeof(hipfftComplex), hipMemcpyHostToDevice));
    callCufft(dev_in, dev_out, n, iflag);
    CUDA_CALL(hipMemcpy(out.data(), dev_out, n * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(dev_in));
    CUDA_CALL(hipFree(dev_out));
    return out;
}