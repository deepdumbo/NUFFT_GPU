#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>
#include <iostream>
#include <stdio.h>

#include "hip/hip_runtime_api.h"
#include "fft_helper.cuh"
#include "gpu_nufft.cuh"

using std::vector;

int MAX_THREADS = 1024;

__global__
void
naiveGriddingKernel(float *dev_x, float *dev_y, hipfftComplex *dev_ftau, float df, float tau,
                    int N, int Mr, int kernelSize) {
    int threadId = threadIdx.x + blockDim.x * blockIdx.x;
    float hx = 2 * HIP_PI_F / Mr;
    // Calculate the kernel
    if (threadId < N) {
        float xi = fmodf(dev_x[threadId] * df, 2.0 * HIP_PI_F);
        int m = 1 + ((int) (xi / hx));
        for (int j = 0; j < kernelSize; ++j) {
            int mmj = -(kernelSize / 2) + j;
            // TODO try __expf
            float kj = expf(-0.25f * powf(xi - hx * (m + mmj), 2) / tau);
            // Assuming Mr > Msp i.e. grid size greater than half of the kernel size
            // TODO modulo instructions are apparently expensive.
            // TODO use reduction for this step instead of atomicAdd.
            int index = (m + mmj + Mr) % Mr;
            atomicAdd(&(dev_ftau[index].x), dev_y[threadId] * kj);
        }
    }
}


__global__
void postProcessingKernel(hipfftComplex *dev_Ftau, hipfftComplex *dev_yt, float* dev_kvec,
        int N, int M, int Mr, float tau) {
    // M/2 threads are needed here.
    int threadId = threadIdx.x + blockDim.x * blockIdx.x;
    float t1x, t1y, t2x, t2y;
    if (threadId < M/2) {
        t1x = dev_Ftau[threadId - M/2 + Mr].x / Mr;
        t1y = dev_Ftau[threadId - M/2 + Mr].y / Mr;
        t2x = dev_Ftau[threadId].x / Mr;
        t2y = dev_Ftau[threadId].y / Mr;
        dev_yt[threadId].x = (1.0 / N) * sqrtf(HIP_PI_F/tau) * expf(tau * powf(dev_kvec[threadId], 2.0)) * t1x;
        dev_yt[threadId].y = (1.0 / N) * sqrtf(HIP_PI_F/tau) * expf(tau * powf(dev_kvec[threadId], 2.0)) * t1y;
        dev_yt[threadId + M/2].x = (1.0 / N) * sqrtf(HIP_PI_F/tau) * expf(tau * powf(dev_kvec[threadId + M/2], 2.0)) * t2x;
        dev_yt[threadId + M/2].y = (1.0 / N) * sqrtf(HIP_PI_F/tau) * expf(tau * powf(dev_kvec[threadId + M/2], 2.0)) * t2y;
    }
}


vector<Complex> nufftGpu(const vector<float> x, const vector<float> y, const int M,
                         const float df, const float eps, const int iflag) {
    std::cout << "Starting GPU NUFFT...\n";
    struct Param param = computeGridParams(M, eps);
    int N = x.size();
    int kernelSize = 2 * param.Msp + 1;
    std::cout << "Gridding kernel has size " << kernelSize << "; grid has size " << param.Mr << ".\n";

    float *dev_x;
    float *dev_y;
    float *dev_kvec;
    hipEvent_t start1, stop1, start2, stop2;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipfftComplex *dev_ftau;
    hipfftComplex *dev_Ftau;
    hipfftComplex *dev_yt;

    // Allocate memory
    CUDA_CALL(hipMalloc((void **) &dev_x, N * sizeof(float)));
    CUDA_CALL(hipMalloc((void **) &dev_y, N * sizeof(float)));
    CUDA_CALL(hipMalloc((void **) &dev_ftau, param.Mr * sizeof(hipfftComplex)));

    // Copy & initialize
    CUDA_CALL(hipMemset(dev_ftau, 0, param.Mr * sizeof(hipfftComplex)));
    CUDA_CALL(hipMemcpy(dev_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dev_y, y.data(), N * sizeof(float), hipMemcpyHostToDevice));

    // Construct the convolved grid
    int blockSize = 1024;
    int gridSize = N / blockSize + 1;
    hipEventRecord(start1);
    naiveGriddingKernel<<<gridSize, blockSize>>>(dev_x, dev_y, dev_ftau, df, param.tau, N, param.Mr, kernelSize);
    hipEventRecord(stop1);
    CUDA_CALL(hipPeekAtLastError());
    CUDA_CALL(hipFree(dev_x));
    CUDA_CALL(hipFree(dev_y));


    // FFT step
    CUDA_CALL(hipMalloc((void **) &dev_Ftau, param.Mr * sizeof(hipfftComplex)));
    callCufft(dev_ftau, dev_Ftau, param.Mr, iflag);
    CUDA_CALL(hipFree(dev_ftau));

    // Reordering and de-gridding.
    CUDA_CALL(hipMalloc((void **) &dev_yt, M * sizeof(hipfftComplex)));
    CUDA_CALL(hipMalloc((void **) &dev_kvec, M * sizeof(hipfftComplex)));
    vector<float> k = getFreq(df, M);
    CUDA_CALL(hipMemcpy(dev_kvec, k.data(), M * sizeof(float), hipMemcpyHostToDevice));


    gridSize = (M / 2 / MAX_THREADS) + 1;
    blockSize = gridSize == 1 ? (M / 2) : MAX_THREADS;
    hipEventRecord(start2);
    postProcessingKernel<<<gridSize, blockSize>>>(dev_Ftau, dev_yt, dev_kvec, N, M, param.Mr, param.tau);
    hipEventRecord(stop2);
    CUDA_CALL(hipPeekAtLastError());
    hipEventSynchronize(stop1);
    hipEventSynchronize(stop2);
    float ms1, ms2;
    hipEventElapsedTime(&ms1, start1, stop1);
    hipEventElapsedTime(&ms2, start2, stop2);
    std::cout << "Gridding kernel took " << ms1 <<  "ms; Post-processing kernel took " << ms2 << "ms.\n";
    vector<Complex> yt = vector<Complex>(M);
    CUDA_CALL(hipMemcpy(yt.data(), dev_yt, M * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(dev_Ftau));
    CUDA_CALL(hipFree(dev_yt));
    CUDA_CALL(hipFree(dev_kvec));
    std::cout << "GPU NUFFT Completed\n";
    return yt;
}


// CUDA GPU Device Query code from the course website.
void queryGpus() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Major revision number: %d\n", prop.major);
        printf("  Minor revision number: %d\n", prop.minor);
        printf("  Total shared memory per block (Bytes): %u\n",  prop.sharedMemPerBlock);
        printf("  Total registers per block: %d\n",  prop.regsPerBlock);
        printf("  Warp size: %d\n",  prop.warpSize);
        printf("  Maximum threads per block: %d\n",  prop.maxThreadsPerBlock);
        printf("  Clock rate (KHz): %d\n",  prop.clockRate);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Total VRAM (Bytes): %u\n",  prop.totalGlobalMem);
        printf("  Total constant memory (Bytes): %u\n",  prop.totalConstMem);
        printf("  Number of SMs: %d\n", prop.multiProcessorCount);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Concurrent copy and execution: %s\n",  (prop.deviceOverlap ? "Yes" : "No"));
        printf("  Concurrent kernels: %s\n",  (prop.concurrentKernels ? "Yes" : "No"));
        printf("  Kernel execution timeout: %s\n",  (prop.kernelExecTimeoutEnabled ? "Yes" : "No"));
    }

}
